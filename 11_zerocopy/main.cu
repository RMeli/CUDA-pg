#include <cassert>
#include <iomanip>
#include <iostream>

#include "dot.h"
#include "mem.h"
#include "num.h"
#include "timing.h"

using namespace std;

int main() {
    // numBlocks should be rather small
    // The last part of the reduction operator is performed on the CPU
    constexpr std::size_t numBlocks{64};
    constexpr std::size_t numThreadsPerBlock{512};

    constexpr std::size_t n{33554432};
    constexpr std::size_t reps{10};

    double x{1.0}, y{2.0};

    Timer t;
    double time{0.0};
    double timemalloch{0.0};
    double timefreeh{0.0};

    CUDATimer ct;
    double ctime{0.0};

    double* x_host{nullptr};
    double* y_host{nullptr};
    double r{0.0};

    hipDeviceProp_t prop;
    int device;
    auto status = hipGetDevice(&device);
    cuda_check_status(status);
    status = hipGetDeviceProperties(&prop, device);
    cuda_check_status(status);

    // Unsure that mapped memory is supported by the device
    if (prop.canMapHostMemory != 1) {
        cout << "Device cannot map memory." << endl;
        return 0;
    } else {
        // Enable CUDA runtime to allocate mapped memory
        hipSetDeviceFlags(hipDeviceMapHost);
    }

    cout << "dot (cpu)..." << endl;
    for (std::size_t i{0}; i < reps; i++) {
        t.start();
        x_host = malloc_host(n, x);
        y_host = malloc_host(n, y);
        timemalloch += t.stop();

        t.start();
        dot_cpu(x_host, y_host, r, n);
        time += t.stop();

        assert(nearly_equal(r, static_cast<double>(2 * n)));

        t.start();
        free_host(x_host);
        free_host(y_host);
        timefreeh += t.stop();
    }
    cout << "  malloc: " << timemalloch << " ms" << endl;
    cout << "  dot: " << time << " ms" << endl;
    cout << "  free: " << timefreeh << " ms" << endl;

    // Reset times
    timemalloch = 0.0;
    timefreeh = 0.0;

    cout << "dot (gpu)..." << endl;
    for (std::size_t i{0}; i < reps; i++) {
        t.start();
        x_host = malloc_host(n, x);
        y_host = malloc_host(n, y);
        timemalloch += t.stop();

        ct.start();
        dot_gpu<numBlocks, numThreadsPerBlock>(x_host, y_host, r, n);
        ctime += ct.stop();

        assert(nearly_equal(r, static_cast<double>(2 * n)));

        t.start();
        free_host(x_host);
        free_host(y_host);
        timefreeh += t.stop();
    }
    cout << "  malloc (host): " << timemalloch << " ms" << endl;
    cout << fixed << setprecision(0) << "  dot: " << ctime << " ms" << endl;
    cout << "  free (host): " << timefreeh << " ms" << endl;

    cout << "dot (gpu | mapped)..." << endl;
    for (std::size_t i{0}; i < reps; i++) {
        t.start();
        x_host = malloc_mapped_readonly(n, x);
        y_host = malloc_mapped_readonly(n, y);
        timemalloch += t.stop();

        ct.start();
        dot_gpu<numBlocks, numThreadsPerBlock>(x_host, y_host, r, n);
        ctime += ct.stop();

        assert(nearly_equal(r, static_cast<double>(2 * n)));

        t.start();
        free_cuhost(x_host);
        free_cuhost(y_host);
        timefreeh += t.stop();
    }
    cout << "  malloc (host): " << timemalloch << " ms" << endl;
    cout << fixed << setprecision(0) << "  dot: " << ctime << " ms" << endl;
    cout << "  free (host): " << timefreeh << " ms" << endl;

    return 0;
}