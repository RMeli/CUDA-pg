#include "hip/hip_runtime.h"
#include <chrono>
#include <stdexcept>

#include "timing.h"
#include "err.h"

using duration = std::chrono::milliseconds;

void Timer::start() {
        ticking = true;

        // Get current time
        ti = std::chrono::high_resolution_clock::now();
}

double Timer::stop() {
        // Get current time
        tf = std::chrono::high_resolution_clock::now();

        // Check if clock was started
        if (!ticking) {
            throw std::runtime_error("Timer not started.");
        }

        ticking = false;

        // Compute elapsed time between start and stop (in milliseconds)
        auto time_ms = std::chrono::duration_cast<duration>(tf - ti);

        return time_ms.count();
    }

    CUDATimer::CUDATimer() {
        // Create start event
        auto status = hipEventCreate(&ti);
        cuda_check_status(status);

        // Create stop event
        status = hipEventCreate(&tf);
        cuda_check_status(status);
    }

    CUDATimer::~CUDATimer() {
        // Destroy start event
        auto status = hipEventDestroy(ti);
        cuda_check_status(status);

        // Destroy stop event
        status = hipEventDestroy(tf);
        cuda_check_status(status);
    }

    void CUDATimer::start() {
        ticking = true;

        // Get current time
        auto status = hipEventRecord(ti, 0);
        cuda_check_status(status);
    }

    double CUDATimer::stop() {
        // Get current time
        auto status = hipEventRecord(tf, 0);
        cuda_check_status(status);

        // Syncronize event
        status = hipEventSynchronize(tf);
        cuda_check_status(status);

        // Check if clock was started
        if (!ticking) {
            throw std::runtime_error("Timer not started.");
        }
        ticking = false;

        // Compute elapsed time between start and stop (in milliseconds)
        float time_ms;
        status = hipEventElapsedTime(&time_ms, ti, tf);
        cuda_check_status(status);

        return time_ms;
    }