
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

void cuda_check_status(hipError_t status) {
    if (status != hipSuccess) {
        std::cerr << "ERROR: " << hipGetErrorString(status) << std::endl;
        exit(1);
    }
}
